#include "hip/hip_runtime.h"

#include "mem_alloc_tp.h"
#include "TP.h"
#define NUM_OBJ 512
class S1 {
public:
  int var;
  __host__ __device__ S1() {}
  virtual __host__ __device__ void inc() = 0;
  virtual __host__ __device__ void dec() = 0;
};

class S2 : public S1 {
public:
  __host__ __device__ S2() {}
  __host__ __device__ void inc() { this->var += 2; }

  __host__ __device__ void dec() { this->var -= 2; }
};

__global__ void kernel(S1 **ptr) {
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  // this variable must be defined in every kerenl that uses COAL
  void **vtable;
  if (tid < NUM_OBJ) {
    S1 * obPtr = ptr[tid];
    TP_S1_inc(obPtr);
    CLEANPTR(obPtr,S1 *)->inc();
  }
}

int main() {

  mem_alloc shared_mem(4ULL * 1024 * 1024 * 1024);
  obj_alloc my_obj_alloc(&shared_mem, 1024 * 1024ULL);
  S1 **ptr = (S1 **)my_obj_alloc.calloc<S1 *>(NUM_OBJ);
  for (int i = 0; i < NUM_OBJ; i++)
    ptr[i] = (S1 *)my_obj_alloc.my_new<S2>();

  printf("Objects Creation Done\n");

  // after we get done with creating the objects
  // we ask the SharedOA to create the vfun Table for TP
  my_obj_alloc.create_table();

  // we get a pointer to vtable
  vfun_table = my_obj_alloc.get_vfun_table();

  // to access the virtual function from device we call toDevice
  my_obj_alloc.toDevice();
  int blockSize = 256;
  int numBlocks = (NUM_OBJ + blockSize - 1) / blockSize;
  kernel<<<numBlocks, blockSize>>>(ptr);
  hipDeviceSynchronize();
  printf("Device Call Done\n");



  for (int i = 0; i < NUM_OBJ; i++)
    printf("ptr[%d].var = %d \n", i,  CLEANPTR(ptr[i],S1 *)->var);

  return 0;
}